#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cnn.h"

#define NUM_THREADS 256
#define RUN_COMPARE false

using namespace std;

__device__ void block_conv(global_config_t * global_config, block_config_t* block_config,
	int * images, int * filters, int * result, 
	int B, int C, int K, int W, int H, int RP, int RPP, int SP, int SPP){

	int b = 0, c = 0, k = 0, w = 0, h = 0, rp = 0, rpp = 0, sp = 0, spp = 0;

	int block_B = min (global_config->B - B, block_config->block_B);
	int block_C = min (global_config->C - C, block_config->block_C);
	int block_K = min (global_config->K - K, block_config->block_K);
	int block_W = min (global_config->W - W, block_config->block_W);
	int block_H = min (global_config->H - H, block_config->block_H);
	int block_Rp = min (global_config->R/global_config->sigW-RP, block_config->block_Rp);
	int block_Rpp = min (global_config->sigW - RPP, block_config->block_Rpp);
	int block_Sp = min (global_config->S/global_config->sigH - SP, block_config->block_Sp);
	int block_Spp = min (global_config->sigH - SPP, block_config->block_Spp);

	for (b = 0;b < block_B; ++b)
	{
		for (c = 0;c < block_C; ++c)
		{
			for (k = 0; k < block_K; ++k)
			{
				for (w = 0; w < block_W; ++w)
				{
					for (h = 0; h < block_H; ++h)
					{
						for (rp = 0 ; rp < block_Rp; ++rp)
						{
							for (rpp = 0 ; rpp < block_Rpp; ++rpp)
							{
								for (sp = 0; sp < block_Sp; ++sp)
								{
									for (spp = 0; spp < block_Spp; ++spp)
									{
                                        //cerr << (k+K)*global_config->H*global_config->W*global_config->B+(h+H)*global_config->W*global_config->B+(w+W)*global_config->B+b+B << " ";
										atomicAdd(result+(k+K)*global_config->H*global_config->W*global_config->B+(h+H)*global_config->W*global_config->B+(w+W)*global_config->B+b+B,
										images[(rpp+RPP+global_config->sigW*(rp+RP+w+W))*((global_config->H-1)*global_config->sigH+global_config->S)*global_config->C*global_config->B
											+ (SPP+spp+global_config->sigH*(sp+SP+h+H))*global_config->C*global_config->B+(c+C)*global_config->B+b+B]
										* filters[(k+K)*global_config->S*global_config->R*global_config->C+(global_config->sigW*(rp+RP)+rpp+RPP)*global_config->S*global_config->C+(global_config->sigH*(sp+SP)+spp+SPP)*global_config->C+c+C]);
									}
								}
							}
						}
					}
				}
			}
		}
	}
}


__global__ void gpu_conv(global_config_t * gpu_config_global, block_config_t* gpu_config_block,
	int * gpu_input, int * gpu_filter, int * gpu_output,
	int input_size, int filter_size, int output_size,
	int nb, int nk, int nw, int nh){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= 256) return;
	int b_steps = gpu_config_global->B / gpu_config_block->block_B / nb;
	int b_index = tid / (256 / nb);
	int b_start = gpu_config_block->block_B * (b_index) * b_steps;

	int k_index = tid / (256 / (nb * nk)) % (nk);
	int k_steps = gpu_config_global->K / gpu_config_block->block_K / nk;
	int k_start = gpu_config_block->block_K * k_index * k_steps;

	int w_index = tid / (256 / (nb * nk * nw)) % nw;
	int w_steps = gpu_config_global->W / gpu_config_block->block_W / nw;
	int w_start = gpu_config_block->block_W * w_index * w_steps;

	int h_index = tid / (256 / (nb * nk * nw * nh)) % nh;
	int h_steps = gpu_config_global->H / gpu_config_block->block_H / nh;
	int h_start = gpu_config_block->block_H * h_index * h_steps;

	int b_step, k_step, w_step, h_step;
	int b = 0, c = 0, k = 0, w = 0, h = 0, rp = 0, rpp = 0, sp = 0, spp = 0;
	for (b = b_start, b_step = b_steps;b < gpu_config_global->B && b_step > 0; b += gpu_config_block->block_B, b_step--)
	{
		for (c = 0;c < gpu_config_global->C; c += gpu_config_block->block_C)
		{
			for (k = k_start, k_step = k_steps; k < gpu_config_global->K && k_step > 0; k += gpu_config_block->block_K, k_step--)
			{
				for (w = w_start, w_step = w_steps; w < gpu_config_global->W && w_step > 0; w += gpu_config_block->block_W, w_step--)
				{
					for (h = h_start, h_step = h_steps; h < gpu_config_global->H && h_step > 0; h += gpu_config_block->block_H, h_step--)
					{
						for (rp = 0; rp < gpu_config_global->R / gpu_config_global->sigW; rp += gpu_config_block->block_Rp)
						{
							for (rpp = 0 ; rpp < gpu_config_global->sigW; rpp += gpu_config_block->block_Rpp)
							{
								for (sp =0 ; sp < gpu_config_global->S / gpu_config_global->sigH; sp += gpu_config_block->block_Sp)
								{
									for (spp = 0; spp < gpu_config_global->sigH; spp += gpu_config_block->block_Spp)
									{
										block_conv(gpu_config_global, gpu_config_block, gpu_input, gpu_filter, gpu_output, b, c, k, w, h, rp, rpp, sp, spp);
									}
								}
							}
						}
					}
				}
			}
		}
	}


}

void compute_conv(global_config_t * global_config, block_config_t* block_config,
	int * images, int * filters, int * result,
	int input_size, int filter_size, int output_size,
	int nb, int nk, int nw, int nh){

	hipDeviceSynchronize();
    int * gpu_input, * gpu_filter, *gpu_output;
    global_config_t * gpu_config_global;
    block_config_t * gpu_config_block;
    if(hipSuccess != hipMalloc((void **) &gpu_config_global, sizeof(global_config_t)))
    	cerr << "gpu global config allocation failed" << endl;
    if(hipSuccess != hipMalloc((void **) &gpu_config_block, sizeof(block_config_t)))
    	cerr << "gpu block config allocation failed" << endl;
    if(hipSuccess != hipMalloc((void **) &gpu_input, input_size * sizeof(int)))
    	cerr << "gpu input allocation failed" << endl;
    if(hipSuccess != hipMalloc((void **) &gpu_filter, filter_size * sizeof(int)))
    	cerr << "gpu filter allocation failed" << endl;
    if(hipSuccess != hipMalloc((void **) &gpu_output, output_size * sizeof(int)))
    	cerr << "gpu output allocation failed" << endl;

    hipMemcpy(gpu_input, images, input_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_filter, filters, filter_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(gpu_output, 0, output_size * sizeof(int));
    hipMemcpy(gpu_config_global, global_config, sizeof(global_config_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_config_block, block_config, sizeof(block_config_t), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	double simulation_time = read_timer();

	gpu_conv<<<1,NUM_THREADS>>>(gpu_config_global, gpu_config_block, gpu_input, gpu_filter, gpu_output, input_size, filter_size, output_size, nb, nk, nw, nh);
	
	hipDeviceSynchronize();
	cout << "GPU CNN: " << read_timer() - simulation_time << endl;

    hipMemcpy(result, gpu_output, output_size * sizeof(int), hipMemcpyDeviceToHost);

	
}

int main(int argc, char **argv)
{
	hipDeviceSynchronize();
// Initialize input image
    srand(time(NULL));

    global_config_t test_global = {256, 512, 512, 6, 3, 6, 6, 2, 2};
    block_config_t test_block = {2, 64, 64, 3, 3, 3, 3, 1, 1};
    // block_config_t test_block = {100, 6, 6, 6, 3, 3, 3, 1, 1};
    // global_config_t test_global = {1, 4, 4, 1, 1, 1, 1, 1, 1};
    // block_config_t test_block = {1, 2, 2, 1, 1, 1, 1, 1, 1};  
    int input_w = test_global.sigW * (test_global.W - 1) + test_global.R;
    int input_h = test_global.sigH * (test_global.H - 1) + test_global.S;

    int input_size = input_h * input_w * test_global.C * test_global.B;
    int output_size = test_global.K * test_global.H * test_global.W * test_global.B;
    int filter_size = test_global.K * test_global.R * test_global.S * test_global.C;

    int * filter = new int[filter_size];

    for (int i = 0; i < filter_size; i++)
    {
        filter[i] = 1;
    }

    int * test_input = new int[input_size];

    for (int i = 0; i < input_size; i++)
    {
        test_input[i] = rand() % 256;
    }

    // int * output_naive = new int[output_size]();
    int * output_gpu = new int[output_size]();

    int nb = read_int( argc, argv, "-nb", 8 );
    int nk = read_int( argc, argv, "-nk", 2 );
    int nw = read_int( argc, argv, "-nw", 4 );
    int nh = read_int( argc, argv, "-nh", 4 );

	compute_conv(&test_global, &test_block, test_input, filter, output_gpu, input_size, filter_size, output_size, nb, nk, nw, nh);
	

	if (RUN_COMPARE){
		int * output_naive = new int[output_size]();

		double simulation_time = read_timer();
		naive_cnn(test_input, filter, output_naive, test_global);
		cout << "Naive CNN: " << read_timer() - simulation_time << endl;
		bool correct = true;
		for (int i = 0; i < output_size; i++)
	    {
	    	if(output_naive[i] != output_gpu[i]){
	    		correct = false;
	        	cout << output_naive[i] << "  VS.  " << output_gpu[i] << endl;
	        	break;
	    	}
	    }

	    if(correct) cout << "All results matches" << endl;

	    delete[] output_naive;
	}
		

    delete[] test_input; 
    delete[] output_gpu;
	return 0;
}
